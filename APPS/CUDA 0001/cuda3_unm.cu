#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Manipulation with prepared image.
//
// ***********************************************************************

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_img.h"


__global__ void kernel_rotate( uchar4 *original,uchar4 *rotate, int sizex, int sizey )
{
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( x >= sizex ) return;
    if ( y >= sizey ) return;
    
    rotate[y * sizex + x] = original[(sizey - y - 1) * sizex + x];
 
}
 
void rotateImage( uchar4 *original, uchar4 *rotated, int width, int height )
{
    hipError_t cerr;
    
    uchar4 *cudaOriginal;
    uchar4 *cudaRotate;
    cerr = hipMalloc( &cudaOriginal, width * height * sizeof( uchar4 ) );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    cerr = hipMalloc( &cudaRotate, width * height * sizeof( uchar4 ) );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
   
    cerr = hipMemcpy( cudaOriginal, original, width * height * sizeof( uchar4 ), hipMemcpyHostToDevice );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    int block = 16;
    dim3 blocks( ( width + block - 1 ) / block, ( height + block - 1 ) / block );
    dim3 threads( block, block );
 
    
    kernel_rotate<<< blocks, threads >>>( cudaOriginal, cudaRotate, width, height );
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
 
   
    cerr = hipMemcpy( rotated, cudaRotate, width * height * sizeof( uchar4 ), hipMemcpyDeviceToHost );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    
    hipFree( cudaRotate );
    hipFree( cudaOriginal );
 
}

__global__ void kernel_resize(uchar3 *original, uchar3 *resize, int sizex, int sizey, int sizex1, int sizey1) 
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
    if ( x >= sizex ) return;
    if ( y >= sizey ) return;
    
	uchar3 pixel1 = original[y*2 * sizex + x*2];
	uchar3 pixel2 = original[(y*2 + 1) * sizex + x*2 + 1];

	resize[y * sizex1 + x].x = (pixel1.x + pixel2.x) / 9;
	resize[y * sizex1 + x].y = (pixel1.y + pixel2.y) / 9;
	resize[y * sizex1 + x].z = (pixel1.z + pixel2.z) / 9;
}

uchar3* resizeImage(uchar3 *img, int sizex, int sizey, int sizex1, int sizey1) 
{
	uchar3 *picture = new uchar3[sizex1 * sizey1];

	uchar3 *original = NULL;
	uchar3 *resize = NULL;
	cerr = hipMalloc(&original, sizeof(uchar3) * sizex * sizey);
	cerr = hipMalloc(&resize, sizeof(uchar3) * sizex1 * sizey1);

	cerr = hipMemcpy(original, img, sizeof(uchar3) * sizex * sizey, hipMemcpyHostToDevice);

	int count = 10;
	dim3 blocks((sizex1 + count)/ count, (sizey1 + count) / count);
	dim3 threads(count, count);
	kernel_resize<<<blocks, threads>>>(original, resize, sizex, sizey, sizex1, sizey1);
	cerr = hipPeekAtLastError();
	cerr = hipMemcpy(picture, resize, sizeof(uchar3)*sizex1*sizey1, hipMemcpyDeviceToHost);
	cerr = hipFree(original);
	cerr = hipFree(resize);

	return picture;
}