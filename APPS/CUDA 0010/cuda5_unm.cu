#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <string.h>

#include "cuda_img.h"

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_text( CudaImg t_color_cuda_img, int2 t_pos, char* t_text, char* t_font, uchar2 t_fsize, uchar3 t_color )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    char l_znak = t_text[blockIdx.x];
    char l_bity = t_font[l_znak * t_fsize.y + threadIdx.y];

    if(l_bity & (1 << threadIdx.x))
        t_color_cuda_img.m_p_uchar3[ (l_y + t_pos.y) * t_color_cuda_img.m_size.x + l_x + t_pos.x ] = t_color;
}

void cu_text( CudaImg t_color_pic, int2 t_pos, const char* t_text, char* t_font, uchar2 t_fsize, uchar3 t_color )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size_x = t_fsize.x;
    int l_block_size_y = t_fsize.y;
    dim3 l_blocks( strlen(t_text), 1 );
    dim3 l_threads( l_block_size_x, l_block_size_y );
    char* l_text;
    hipMallocManaged(&l_text, strlen(t_text));
    strcpy(l_text, t_text);
    kernel_text<<< l_blocks, l_threads >>>( t_color_pic, t_pos, l_text, t_font, t_fsize, t_color );
    hipFree(l_text);

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

__global__ void kernel_rotate( uchar4 *original,uchar4 *rotate, int sizex, int sizey )
{
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( x >= sizex ) return;
    if ( y >= sizey ) return;
    
    rotate[y * sizex + x] = original[(sizey - y - 1) * sizex + x];
 
}
 
void rotateImage( uchar4 *original, uchar4 *rotated, int width, int height )
{
    hipError_t cerr;
    
    uchar4 *cudaOriginal;
    uchar4 *cudaRotate;
    cerr = hipMalloc( &cudaOriginal, width * height * sizeof( uchar4 ) );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    cerr = hipMalloc( &cudaRotate, width * height * sizeof( uchar4 ) );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
   
    cerr = hipMemcpy( cudaOriginal, original, width * height * sizeof( uchar4 ), hipMemcpyHostToDevice );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    int block = 16;
    dim3 blocks( ( width + block - 1 ) / block, ( height + block - 1 ) / block );
    dim3 threads( block, block );
 
    
    kernel_rotate<<< blocks, threads >>>( cudaOriginal, cudaRotate, width, height );
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
 
   
    cerr = hipMemcpy( rotated, cudaRotate, width * height * sizeof( uchar4 ), hipMemcpyDeviceToHost );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    
    hipFree( cudaRotate );
    hipFree( cudaOriginal );
 
}



// Demo kernel to create chess board
__global__ void kernel_chessboard( CudaImg t_color_cuda_img )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    unsigned char b_or_w = 255 * ( ( blockIdx.x + blockIdx.y ) & 1 );

    // Store point into image
    t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ] = { b_or_w, b_or_w, b_or_w };
}

void cu_create_chessboard( CudaImg t_color_cuda_img, int t_square_size )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + t_square_size - 1 ) / t_square_size,
                   ( t_color_cuda_img.m_size.y + t_square_size - 1 ) / t_square_size );
    dim3 l_threads( t_square_size, t_square_size );
    kernel_chessboard<<< l_blocks, l_threads >>>( t_color_cuda_img );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_alphaimg( CudaImg t_color_cuda_img, uchar3 t_color )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    int l_diagonal = sqrtf( t_color_cuda_img.m_size.x * t_color_cuda_img.m_size.x + t_color_cuda_img.m_size.y * t_color_cuda_img.m_size.y );
    int l_dx = l_x - t_color_cuda_img.m_size.x / 2;
    int l_dy = l_y - t_color_cuda_img.m_size.y / 2;
    int l_dxy = sqrtf( l_dx * l_dx + l_dy * l_dy ) - l_diagonal / 2;

    // Store point into image
    t_color_cuda_img.m_p_uchar4[ l_y * t_color_cuda_img.m_size.x + l_x ] =
        { t_color.x, t_color.y, t_color.z, ( unsigned char ) ( 255 - 255 * l_dxy / ( l_diagonal / 2 ) ) };
}

void cu_create_alphaimg( CudaImg t_color_cuda_img, uchar3 t_color )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 32;
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( t_color_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_alphaimg<<< l_blocks, l_threads >>>( t_color_cuda_img, t_color );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaImg t_big_cuda_pic, CudaImg t_small_cuda_pic, int2 t_position )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_small_cuda_pic.m_size.y ) return;
    if ( l_x >= t_small_cuda_pic.m_size.x ) return;
    int l_by = l_y + t_position.y;
    int l_bx = l_x + t_position.x;
    if ( l_by >= t_big_cuda_pic.m_size.y || l_by < 0 ) return;
    if ( l_bx >= t_big_cuda_pic.m_size.x || l_bx < 0 ) return;

    // Get point from small image
    uchar4 l_fg_bgra = t_small_cuda_pic.m_p_uchar4[ l_y * t_small_cuda_pic.m_size.x + l_x ];
    uchar3 l_bg_bgr = t_big_cuda_pic.m_p_uchar3[ l_by * t_big_cuda_pic.m_size.x + l_bx ];
    uchar3 l_bgr = { 0, 0, 0 };

    // compose point from small and big image according alpha channel
    l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
    l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
    l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

    // Store point into image
    t_big_cuda_pic.m_p_uchar3[ l_by * t_big_cuda_pic.m_size.x + l_bx ] = l_bgr;
}

void cu_insertimage( CudaImg t_big_cuda_pic, CudaImg t_small_cuda_pic, int2 t_position )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 32;
    dim3 l_blocks( ( t_small_cuda_pic.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( t_small_cuda_pic.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_insertimage<<< l_blocks, l_threads >>>( t_big_cuda_pic, t_small_cuda_pic, t_position );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}
