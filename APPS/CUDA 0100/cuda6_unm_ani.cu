#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Simple animation.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "cuda_img.h"
#include "animation.h"


__global__ void kernel_rotate( uchar4 *original,uchar4 *rotate, int sizex, int sizey )
{
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( x >= sizex ) return;
    if ( y >= sizey ) return;
    
    rotate[y * sizex + x] = original[(sizey - y - 1) * sizex + x];
 
}
 
void cu_rotate( uchar4 *original, uchar4 *rotated, int width, int height )
{
    hipError_t cerr;
    
    uchar4 *cudaOriginal;
    uchar4 *cudaRotate;
    cerr = hipMalloc( &cudaOriginal, width * height * sizeof( uchar4 ) );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    cerr = hipMalloc( &cudaRotate, width * height * sizeof( uchar4 ) );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
   
    cerr = hipMemcpy( cudaOriginal, original, width * height * sizeof( uchar4 ), hipMemcpyHostToDevice );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    int block = 16;
    dim3 blocks( ( width + block - 1 ) / block, ( height + block - 1 ) / block );
    dim3 threads( block, block );
 
    
    kernel_rotate<<< blocks, threads >>>( cudaOriginal, cudaRotate, width, height );
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
 
   
    cerr = hipMemcpy( rotated, cudaRotate, width * height * sizeof( uchar4 ), hipMemcpyDeviceToHost );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );    
 
    
    hipFree( cudaRotate );
    hipFree( cudaOriginal );
 
}


__global__ void rotate_ok(uchar3 *original, uchar3 *rotated, int width, int height) 
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= width || y >= height) 
    {
		return;
	}


	int sizex = width / 2;
	int sizey = height / 2;

	float theta = 30 * 3.14 / 180;
	int x2 = (x-sizex) * cos(theta) - (y - sizey) * sin(theta) + sizex;
	int y2 = (x-sizex) * sin(theta) + (y - sizey) * cos(theta) + sizey;

	if(x2 >= 0 && x2 < width && y2 >=0 && y2 < height) 
    {
		rotated[y * width + x] = original[y2 * width + x2];
	} 
    else 
    {
		rotated[y * width + x] = (uchar3) {0, 0, 0};
	}
}

uchar3* cu_rotate_ok(uchar3 *img, int width, int height) 
{
	uchar3 *picture = new uchar3[width * heigth];

	uchar3 *original = NULL;
	uchar3 *rotated = NULL;
	cerr = hipMalloc(&original, sizeof(uchar3) * width * height);
	cerr = hipMalloc(&rotated, sizeof(uchar3) * width * height);

	cerr = hipMemcpy(original, img, sizeof(uchar3) * width * height, hipMemcpyHostToDevice);

	int count = 10;
	dim3 blocks((width + count)/ count, (height + count) / count);
	dim3 threads(count, count);
	rotate_ok<<<blocks, threads>>>(original, rotated, width, height);
	cerr = hipPeekAtLastError();
	cerr = hipMemcpy(picture, rotated, sizeof(uchar3) * width * height, hipMemcpyDeviceToHost);
	cerr = hipFree(original);
	cerr = hipFree(rotated);

	return picture;
}


__global__ void kernel_blur( uchar3 *original, uchar3 *blurred, float t_level )
{
    for ( blockDim.x = 1; blockDim.x < threadIdx.y - 1; blockDim.x++ )
        for ( blockDim.y = 1; blockDim.y < threadIdx.x - 1; blockDim.y++ )
        {
            // initialize sum
            uchar3 l_bgr32 = { 0, 0, 0 };
            // loop for all neighbours
            for ( int nx = -1; nx <= 1; nx++ )
                for( int ny = -1; ny <= 1; ny++ )
                {
                    // pickup point from orig figure
                    uchar3 l_bgr = original(blockDim.y + ny, blockDim.x + nx);
                    // sum of r/g/b colors
                    for ( int b = 0; b < 3; b++ )
                    {
                        if ( !nx && !ny ) 
                            l_bgr32[ b ] += l_bgr[ b ];  
                        else 
                            l_bgr32[ b ] += l_bgr[ b ] * t_level;
                    }
                }
            // average
            l_bgr32 /= 1 + 8 * t_level;
            // put pixel into blur image
            blurred(blockDim.y, blockDim.x) = l_bgr32;
        }
}

void cu_blur( uchar3 *original, uchar3 *blurred, float t_level )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_cuda_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_cuda_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_blur<<< l_blocks, l_threads >>>( *original, *blurred, t_level);

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

__global__ void kernel_insert( CudaImg big_img, CudaImg small_img, int2 t_position )
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= small_img.m_size.y ) return;
    if ( l_x >= small_img.m_size.x ) return;
    int l_by = l_y + t_position.y;
    int l_bx = l_x + t_position.x;
    if ( l_by >= big_img_size.y  l_by < 0 ) return;
    if ( l_bx >= big_img.m_size.x  l_bx < 0 ) return;

    big_img.m_p_uchar3[ l_by * big_img.m_size.x + l_bx ] = small_img.m_p_uchar3[ l_y * small_img.m_size.x + l_x ];
}

__global__ void kernel_clear( CudaImg img )
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    img.m_p_uchar3[ l_y * img.m_size.x + l_x ] = { 0, 0, 0 };
}

void cu_insert( CudaImg big_img, CudaImg small_img, int2 t_position )
{
    hipError_t l_cerr;

    int l_block_size = 32;
    dim3 l_blocks( ( small_img.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( small_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );

    dim3 l_blocks_b( ( big_img.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( big_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads_b( l_block_size, l_block_size );

    kernel_clear<<<l_blocks_b, l_threads_b>>>(img);
    kernel_insert<<< l_blocks, l_threads >>>( big_img, small_img, t_position );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", LINE, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

void Animation::start( uchar4 *original,uchar4 *rotate, int sizex, int sizey )
{
	if ( m_initialized ) return;
	hipError_t l_cerr;

	m_bg_cuda_img = t_bg_cuda_img;
	m_res_cuda_img = t_bg_cuda_img;
	m_ins_cuda_img = t_ins_cuda_img;

	// Memory allocation in GPU device
	// Memory for background
	l_cerr = hipMalloc( &m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof( uchar3 ) );
	if ( l_cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	// Creation of background gradient
	int l_block_size = 32;
	dim3 l_blocks( ( m_bg_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( m_bg_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_rotate<<< l_blocks, l_threads >>>(*original, *rotate, sizex, sizey);

	m_initialized = 1;
}

void Animation::next( uchar3 *original, uchar3 *blurred, float t_level );
{
	if ( !m_initialized ) return;

	hipError_t cerr;

	// Copy data internally GPU from background into result
	cerr = hipMemcpy( m_res_cuda_img.m_p_void, m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// insert picture
	int l_block_size = 32;
	dim3 l_blocks( ( m_ins_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( m_ins_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
    kernel_blur<<< l_blocks, l_threads >>>(*original, *blurred, t_level );

	// Copy data to GPU device
	cerr = hipMemcpy( t_res_cuda_img.m_p_void, m_res_cuda_img.m_p_void, m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

}

void Animation::stop()
{
	if ( !m_initialized ) return;

	hipFree( m_bg_cuda_img.m_p_void );
	hipFree( m_res_cuda_img.m_p_void );
	hipFree( m_ins_cuda_img.m_p_void );

	m_initialized = 0;
}


