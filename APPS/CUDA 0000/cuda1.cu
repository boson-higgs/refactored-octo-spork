#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage.
// Global variables usage in threads, the use of printf.
//
// Every thread displays information of its position in block,
// position of block in grid and global position.
//
// ***********************************************************************


#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void insert_picture_half( CudaPic t_color_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	// Get point from color picture
	uchar3 l_bgr = t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ];

	t_color_pic.at3(l_x, l_y).x = l_bgr.x / 2;
	t_color_pic.at3(l_x, l_y).y = l_bgr.y / 2;
	t_color_pic.at3(l_x, l_y).z = l_bgr.z / 2;
}



void cu_run_insert_picture_half( CudaPic t_color_pic)
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 16;
	dim3 l_blocks( ( t_color_pic.m_size.x + l_block_size - 1 ) / l_block_size, ( t_color_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insert_picture_half<<< l_blocks, l_threads >>>( t_color_pic);

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}


__global__ void insert_picture( CudaPic t_color_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	// Get point from color picture
	uchar3 l_bgr = t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ];

    if(t_color_pic.at3(l_x, l_y).x > 127)
    {
        t_color_pic.at3(l_x, l_y).x = 127;   
    }
	
    if(t_color_pic.at3(l_x, l_y).y > 127)
    {
        t_color_pic.at3(l_x, l_y).y = 127;
    }
	
    if(t_color_pic.at3(l_x, l_y).z > 127)
    {
        t_color_pic.at3(l_x, l_y).z = 127;
    }
}



void cu_run_insert_picture( CudaPic t_color_pic)
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 16;
	dim3 l_blocks( ( t_color_pic.m_size.x + l_block_size - 1 ) / l_block_size, ( t_color_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insert_picture<<< l_blocks, l_threads >>>( t_color_pic);

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}