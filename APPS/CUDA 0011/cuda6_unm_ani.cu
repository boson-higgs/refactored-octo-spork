#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Simple animation.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "cuda_img.h"
#include "animation.h"




__global__ void kernel_blur( uchar3 *original, uchar3 *blurred, float t_level )
{
    for ( blockDim.x = 1; blockDim.x < threadIdx.y - 1; blockDim.x++ )
        for ( blockDim.y = 1; blockDim.y < threadIdx.x - 1; blockDim.y++ )
        {
            // initialize sum
            uchar3 l_bgr32 = { 0, 0, 0 };
            // loop for all neighbours
            for ( int nx = -1; nx <= 1; nx++ )
                for( int ny = -1; ny <= 1; ny++ )
                {
                    // pickup point from orig figure
                    uchar3 l_bgr = original(blockDim.y + ny, blockDim.x + nx);
                    // sum of r/g/b colors
                    for ( int b = 0; b < 3; b++ )
                    {
                        if ( !nx && !ny ) 
                            l_bgr32[ b ] += l_bgr[ b ];  
                        else 
                            l_bgr32[ b ] += l_bgr[ b ] * t_level;
                    }
                }
            // average
            l_bgr32 /= 1 + 8 * t_level;
            // put pixel into blur image
            blurred(blockDim.y, blockDim.x) = l_bgr32;
        }
}

void cu_blur( uchar3 *original, uchar3 *blurred, float t_level )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_cuda_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_cuda_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_blur<<< l_blocks, l_threads >>>( *original, *blurred, t_level);

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

__global__ void kernel_cv_run_bilin_scale( uchar3 *original, uchar3 *resized )
{
    float l_scale_x = original.blockDim.y - 1;
    float l_scale_y = original.blockDim.x- 1;
    l_scale_x /= resized.blockDim.x;
    l_scale_y /= resized.blockDim.y;

    for ( int l_resize_x = 0; l_resize_x < resized.blockDim.y; l_resize_x++ )
    {
        for ( int l_resize_y = 0; l_resize_y < resized.blockDim.x; l_resize_y++ )
        {
            // new real position
            float l_orig_x = l_resize_x * l_scale_x;
            float l_orig_y = l_resize_y * l_scale_y;
            // diff x and y
            float l_diff_x = l_orig_x - ( int ) l_orig_x;
            float l_diff_y = l_orig_y - ( int ) l_orig_y;

            // points
            uchar3 bgr00 = original(( int ) l_orig_y, ( int ) l_orig_x );
            uchar3 bgr01 = original(( int ) l_orig_y, 1 + ( int ) l_orig_x );
            uchar3 bgr10 = original( 1 + ( int ) l_orig_y, ( int ) l_orig_x );
            uchar3 bgr11 = original( 1 + ( int ) l_orig_y, 1 + ( int ) l_orig_x );

            uchar3 bgr;
            for ( int i = 0; i < 3; i++ )
            {
                // color calculation
                bgr[ i ] = bgr00[ i ] * ( 1 - l_diff_y ) * ( 1 - l_diff_x ) +
                           bgr01[ i ] * ( 1 - l_diff_y ) * ( l_diff_x ) +
                           bgr10[ i ] * ( l_diff_y ) * ( 1 - l_diff_x ) +
                           bgr11[ i ] * ( l_diff_y ) * ( l_diff_x );
                resized( l_resize_y, l_resize_x ) = bgr;
            }
        }
    }
}

void cu_cv_run_bilin_scale( uchar3 *original, uchar3 *resized )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_cuda_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_cuda_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_cv_run_bilin_scale<<< l_blocks, l_threads >>>( *original, *resized );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}


__global__ void rotate_ok(uchar3 *original, uchar3 *rotated, int width, int height) 
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= width || y >= height) 
    {
		return;
	}


	int sizex = width / 2;
	int sizey = height / 2;

	float theta = 30 * 3.14 / 180;
	int x2 = (x-sizex) * cos(theta) - (y - sizey) * sin(theta) + sizex;
	int y2 = (x-sizex) * sin(theta) + (y - sizey) * cos(theta) + sizey;

	if(x2 >= 0 && x2 < width && y2 >=0 && y2 < height) 
    {
		rotated[y * width + x] = original[y2 * width + x2];
	} 
    else 
    {
		rotated[y * width + x] = (uchar3) {0, 0, 0};
	}
}

uchar3* cu_rotate_ok(uchar3 *img, int width, int height) 
{
	uchar3 *picture = new uchar3[width * heigth];

	uchar3 *original = NULL;
	uchar3 *rotated = NULL;
	cerr = hipMalloc(&original, sizeof(uchar3) * width * height);
	cerr = hipMalloc(&rotated, sizeof(uchar3) * width * height);

	cerr = hipMemcpy(original, img, sizeof(uchar3) * width * height, hipMemcpyHostToDevice);

	int count = 10;
	dim3 blocks((width + count)/ count, (height + count) / count);
	dim3 threads(count, count);
	rotate_ok<<<blocks, threads>>>(original, rotated, width, height);
	cerr = hipPeekAtLastError();
	cerr = hipMemcpy(picture, rotated, sizeof(uchar3) * width * height, hipMemcpyDeviceToHost);
	cerr = hipFree(original);
	cerr = hipFree(rotated);

	return picture;
}


__global__ void kernel_text( CudaImg t_color_cuda_img, int2 t_pos, char* t_text, char* t_font, uchar2 t_fsize, uchar3 t_color )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    char l_znak = t_text[blockIdx.x];
    char l_bity = t_font[l_znak * t_fsize.y + threadIdx.y];

    if(l_bity & (1 << threadIdx.x))
        t_color_cuda_img.m_p_uchar3[ (l_y + t_pos.y) * t_color_cuda_img.m_size.x + l_x + t_pos.x ] = t_color;
}

void cu_text( CudaImg t_color_pic, int2 t_pos, const char* t_text, char* t_font, uchar2 t_fsize, uchar3 t_color )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size_x = t_fsize.x;
    int l_block_size_y = t_fsize.y;
    dim3 l_blocks( strlen(t_text), 1 );
    dim3 l_threads( l_block_size_x, l_block_size_y );
    char* l_text;
    hipMallocManaged(&l_text, strlen(t_text));
    strcpy(l_text, t_text);
    kernel_text<<< l_blocks, l_threads >>>( t_color_pic, t_pos, l_text, t_font, t_fsize, t_color );
    hipFree(l_text);

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}



// Demo kernel to create chess board
__global__ void kernel_creategradient( CudaImg t_color_cuda_img )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_cuda_img.m_size.y ) return;
	if ( l_x >= t_color_cuda_img.m_size.x ) return;

	int l_dy = l_x * t_color_cuda_img.m_size.y / t_color_cuda_img.m_size.x + l_y - t_color_cuda_img.m_size.y;
	unsigned char l_color = 255 * abs( l_dy ) / t_color_cuda_img.m_size.y;

	uchar3 l_bgr = ( l_dy < 0 ) ? ( uchar3 ) { l_color, 255 - l_color, 0 } : ( uchar3 ) { 0, 255 - l_color, l_color };

	// Store point into image
	t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ] = l_bgr;
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaImg t_big_cuda_img, CudaImg t_small_cuda_pic, int2 t_position )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_small_cuda_pic.m_size.y ) return;
	if ( l_x >= t_small_cuda_pic.m_size.x ) return;
	int l_by = l_y + t_position.y;
	int l_bx = l_x + t_position.x;
	if ( l_by >= t_big_cuda_img.m_size.y || l_by < 0 ) return;
	if ( l_bx >= t_big_cuda_img.m_size.x || l_bx < 0 ) return;

	// Get point from small image
	uchar4 l_fg_bgra = t_small_cuda_pic.m_p_uchar4[ l_y * t_small_cuda_pic.m_size.x + l_x ];
	uchar3 l_bg_bgr = t_big_cuda_img.m_p_uchar3[ l_by * t_big_cuda_img.m_size.x + l_bx ];
	uchar3 l_bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
	l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

	// Store point into image
	t_big_cuda_img.m_p_uchar3[ l_by * t_big_cuda_img.m_size.x + l_bx ] = l_bgr;
}

void cu_insertimage( CudaImg t_big_cuda_img, CudaImg t_small_cuda_pic, int2 t_position )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_cuda_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_cuda_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( t_big_cuda_img, t_small_cuda_pic, t_position );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

void Animation::start( CudaImg t_bg_cuda_img, CudaImg t_ins_cuda_img )
{
	if ( m_initialized ) return;
	hipError_t l_cerr;

	m_bg_cuda_img = t_bg_cuda_img;
	m_res_cuda_img = t_bg_cuda_img;
	m_ins_cuda_img = t_ins_cuda_img;

	// Memory allocation in GPU device
	// Memory for background
	l_cerr = hipMalloc( &m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof( uchar3 ) );
	if ( l_cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	// Creation of background gradient
	int l_block_size = 32;
	dim3 l_blocks( ( m_bg_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( m_bg_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_creategradient<<< l_blocks, l_threads >>>( m_bg_cuda_img );

	m_initialized = 1;
}

void Animation::next( CudaImg t_res_cuda_img, int2 t_position )
{
	if ( !m_initialized ) return;

	hipError_t cerr;

	// Copy data internally GPU from background into result
	cerr = hipMemcpy( m_res_cuda_img.m_p_void, m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// insert picture
	int l_block_size = 32;
	dim3 l_blocks( ( m_ins_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( m_ins_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( m_res_cuda_img, m_ins_cuda_img, t_position );

	// Copy data to GPU device
	cerr = hipMemcpy( t_res_cuda_img.m_p_void, m_res_cuda_img.m_p_void, m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

}

void Animation::stop()
{
	if ( !m_initialized ) return;

	hipFree( m_bg_cuda_img.m_p_void );
	hipFree( m_res_cuda_img.m_p_void );
	hipFree( m_ins_cuda_img.m_p_void );

	m_initialized = 0;
}


